#include "hip/hip_runtime.h"
#include <dcgn/dcgn.h>
#include <dcgn/CUDAFunctions.h>
#include <cstdlib>
#include <cstdio>

const int MIN_SIZE = 1;
const int MAX_SIZE = 1048576;
const int ITERS = 30;

__global__ void kernel(void * gmem, const dcgn::GPUInitRequest libParam)
{
  dcgn::CommStatus stat;
  dcgn::gpu::init(libParam);
  for (int i = MIN_SIZE; i <= MAX_SIZE; i *= 2)
  {
    dcgn::gpu::barrier(0);
    for (int j = 0; j < ITERS; ++j)
    {
      dcgn::gpu::recv(0, 0, gmem, i, &stat);
    }
    dcgn::gpu::barrier(0);
  }
}

__host__ void gpuKernel(void * info, const dcgn::GPUInitRequest libParam, const uint3 & gridSize, const uint3 & blockSize, const int sharedMemSize, hipStream_t * const stream)
{
  void ** mem = (void ** )info;
  hipMalloc(mem, MAX_SIZE);
  kernel<<<gridSize, blockSize, sharedMemSize, *stream>>>(*mem, libParam);
}

__host__ void gpuDtor(void * info)
{
  hipFree(*(void ** )info);
}

void cpuKernel(void * info)
{
  void * mem = (void * )malloc(MAX_SIZE);
  for (int i = MIN_SIZE; i <= MAX_SIZE; i *= 2)
  {
    dcgn::barrier();
    double t = dcgn::wallTime();
    for (int j = 0; j < ITERS; ++j)
    {
      dcgn::send(1, mem, i);
    }
    dcgn::barrier();
    t = dcgn::wallTime() - t;
    printf("%20.10f\n", t / ITERS);
  }
  free(mem);
}

int main(int argc, char ** argv)
{
  void * gpuMem;
  int gpus[] = { 0, -1 };
  uint3 gs = { 1, 1, 1 }, bs = { 1, 1, 1 };

  dcgn::init(&argc, &argv);
  dcgn::initComm(-1);
  dcgn::initCPU(1);
  dcgn::initGPU(gpus, 1, 0);
  dcgn::start();

  dcgn::launchCPUKernel(0, cpuKernel, 0);
  dcgn::launchGPUKernel(0, gpuKernel, gpuDtor, &gpuMem, gs, bs);

  dcgn::finalize();
  return 0;
}
